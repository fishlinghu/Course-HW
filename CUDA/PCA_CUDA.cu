#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <map>

#define num_of_threads 128


using namespace std;

struct timespec start_time;
struct timespec compute_scatter;
struct timespec before_jacobi;
struct timespec after_jacobi;
struct timespec end_time;

void runTest( int argc, char** argv);


////////////////////////////////////////////////////////////////////////////////
// declaration, forward

//��U�Ӫ��D�S�x�ȯS�x�V�q
/**
* @brief �D???�x?���S���ȤίS���V�q�����J��k
* �Q�ζ����(Jacobi)��k�D???�x?�������S���ȤίS���V�q
* @param pMatrix                n*n matrix, used to store scatter matrix
* @param nDim                   dimension of the matrix
* @param pdblVects              n*n matrix, used to store eigenvectors, column major
* @param dbEps                  maximum error
* @param nJt                    maximum number of iterations
* @param pdbEigenValues         �S����??
* @return
*/
bool Jacobi(double *pMatrix, int nDim, double *pdblVects, double *pdbEigenValues, double dbEps, int nJt)
    {
    for(int i = 0; i < nDim; i ++)
        {
        pdblVects[i*nDim+i] = 1.0f;
        for(int j = 0; j < nDim; j ++)
            {
            if(i != j)
                pdblVects[i*nDim+j]=0.0f;
            }
        }

    int nCount = 0;     //number of iterations
    while(1)
        {
        //Find the biggest element from pMatrix (except diagonal elements)
        double dbMax = pMatrix[1];
        int nRow = 0;
        int nCol = 1;
        for (int i = 0; i < nDim; i ++)          //row
            {
            for (int j = 0; j < nDim; j ++)      //col
                {
                double d = fabs(pMatrix[i*nDim+j]);

                if((i!=j) && (d> dbMax))
                    {
                    dbMax = d;
                    nRow = i;
                    nCol = j;
                    }
                }
            }

        if(dbMax < dbEps)      //check the error
            break;

        if(nCount > nJt)       //number of iterations exceeds nJt
            break;

        nCount++;

        double dbApp = pMatrix[nRow*nDim+nRow];
        double dbApq = pMatrix[nRow*nDim+nCol];
        double dbAqq = pMatrix[nCol*nDim+nCol];

        //compute the rotation angle
        double dbAngle = 0.5*atan2(-2*dbApq,dbAqq-dbApp);
        double dbSinTheta = sin(dbAngle);
        double dbCosTheta = cos(dbAngle);
        double dbSin2Theta = sin(2*dbAngle);
        double dbCos2Theta = cos(2*dbAngle);

        pMatrix[nRow*nDim+nRow] = dbApp*dbCosTheta*dbCosTheta + dbAqq*dbSinTheta*dbSinTheta + 2*dbApq*dbCosTheta*dbSinTheta;
        pMatrix[nCol*nDim+nCol] = dbApp*dbSinTheta*dbSinTheta + dbAqq*dbCosTheta*dbCosTheta - 2*dbApq*dbCosTheta*dbSinTheta;
        pMatrix[nRow*nDim+nCol] = 0.5*(dbAqq-dbApp)*dbSin2Theta + dbApq*dbCos2Theta;
        pMatrix[nCol*nDim+nRow] = pMatrix[nRow*nDim+nCol];

        for(int i = 0; i < nDim; i ++)
            {
            if((i!=nCol) && (i!=nRow))
                {
                int u = i*nDim + nRow;  //p
                int w = i*nDim + nCol;  //q
                dbMax = pMatrix[u];
                pMatrix[u]= pMatrix[w]*dbSinTheta + dbMax*dbCosTheta;
                pMatrix[w]= pMatrix[w]*dbCosTheta - dbMax*dbSinTheta;
                }
            }

        for (int j = 0; j < nDim; j ++)
            {
            if((j!=nCol) && (j!=nRow))
                {
                int u = nRow*nDim + j;  //p
                int w = nCol*nDim + j;  //q
                dbMax = pMatrix[u];
                pMatrix[u]= pMatrix[w]*dbSinTheta + dbMax*dbCosTheta;
                pMatrix[w]= pMatrix[w]*dbCosTheta - dbMax*dbSinTheta;
                }
            }

        //compute eigenvector
        for(int i = 0; i < nDim; i ++)
            {
            int u = i*nDim + nRow;      //p
            int w = i*nDim + nCol;      //q
            dbMax = pdblVects[u];
            pdblVects[u] = pdblVects[w]*dbSinTheta + dbMax*dbCosTheta;
            pdblVects[w] = pdblVects[w]*dbCosTheta - dbMax*dbSinTheta;
            }

        }
    //cout << "HI I am here" << endl;
    //sort eigenvalues and rearrange eigenvectors
    //eigenvalue = elements of the diagonal of pMatrix
	map<double,int> mapEigen;
    for(int i = 0; i < nDim; i ++)
        {
        pdbEigenValues[i] = pMatrix[i*nDim+i];

        mapEigen.insert(make_pair( pdbEigenValues[i],i ) );
        }
    //cout << "1" << endl;
    double *pdbTmpVec = new double[nDim*nDim];
    //cout << "Allocate the fucking memory" << endl;
    std::map<double,int>::reverse_iterator iter = mapEigen.rbegin();
    for (int j = 0; iter != mapEigen.rend(),j < nDim; ++iter,++j)
        {
        for (int i = 0; i < nDim; i ++)
            {
            pdbTmpVec[i*nDim+j] = pdblVects[i*nDim + iter->second];
            }

        //sort eigenvalues
        pdbEigenValues[j] = iter->first;
        }
    //cout << "Set +-" << endl;
    //set +-
    for(int i = 0; i < nDim; i ++)
        {
        double dSumVec = 0;
        for(int j = 0; j < nDim; j ++)
            dSumVec += pdbTmpVec[j * nDim + i];
        if(dSumVec<0)
            {
            for(int j = 0;j < nDim; j ++)
                pdbTmpVec[j * nDim + i] *= -1;
            }
        }
    //cout << "Almost there" << endl;

    memcpy(pdblVects,pdbTmpVec,sizeof(double)*nDim*nDim);
    delete []pdbTmpVec;

    return 1;
    }




double gettime()
    {
    struct timeval t;
    gettimeofday(&t,NULL);
    return t.tv_sec+t.tv_usec*1e-6;
    }

////////////////////////////////////////////////////////////////////////////////
// CUDA kernel
////////////////////////////////////////////////////////////////////////////////
__global__ static void kernel_compute_scatter_matrix_seq(double *d_scatter, const double *d_mean, const double *d_samples, const int workload, const int dim, const int n)
    {
    int blkID = blockIdx.x;
    int trdID = blkID * blockDim.x + threadIdx.x;
    int i, j, row_element_index;
    double total;
    row_element_index = threadIdx.x * workload; // Index of first element

    i = 0;
    while(i < workload)
        {
        if(row_element_index >= dim)
            break;
        else
            {
            //printf("(%d, %d)\n", blkID, row_element_index);
            total = 0;
            j = 0;
            while(j < n)
                {
                total = total + (d_samples[j * dim + blkID] - d_mean[blkID]) * (d_samples[j * dim + row_element_index] - d_mean[row_element_index]);
                ++j;
                }
            d_scatter[blkID * dim + row_element_index] = total;
            }
        ++row_element_index;
        ++i;
        }
    }

__global__ static void kernel_compute_scatter_matrix_seq_memory_sharing(double *d_scatter, const double *d_mean, const double *d_samples, const int workload, const int dim, const int n)
    {
    extern __shared__ double data[];
    int blkID = blockIdx.x;
    int trdID = blkID * blockDim.x + threadIdx.x;
    int i, j, row_element_index;
    double total;
    row_element_index = threadIdx.x * workload; // Index of first element
    i = threadIdx.x;
    while(i < n)
        {
        data[i] = d_samples[i * dim + blkID];
        i = i + num_of_threads;
        }

    __syncthreads();

    i = 0;
    while(i < workload)
        {
        if(row_element_index >= dim)
            break;
        else
            {
            //printf("(%d, %d)\n", blkID, row_element_index);
            total = 0;
            j = 0;
            while(j < n)
                {
                total = total + (data[j] - d_mean[blkID]) * (d_samples[j * dim + row_element_index] - d_mean[row_element_index]);
                ++j;
                }
            d_scatter[blkID * dim + row_element_index] = total;
            }
        ++row_element_index;
        ++i;
        }
    }

__global__ static void kernel_compute_scatter_matrix_interleve(double *d_scatter, const double *d_mean, const double *d_samples, const int workload, const int dim, const int n)
    {
    int blkID = blockIdx.x;
    int trdID = blkID * blockDim.x + threadIdx.x;
    int i, j, row_element_index;
    double total;
    //row_element_index = threadIdx.x * workload;

    i = threadIdx.x;
    while(i < dim)
        {
        j = 0;
        total = 0;
        while(j < n)
            {
            total = total + (d_samples[j * dim + blkID] - d_mean[blkID]) * (d_samples[j * dim + i] - d_mean[i]);
            ++j;
            }
        d_scatter[blkID * dim + i] = total;
        i = i + num_of_threads;
        }
    }

__global__ static void kernel_compute_scatter_matrix_interleve_memory_sharing(double *d_scatter, const double *d_mean, const double *d_samples, const int workload, const int dim, const int n)
    {
    extern __shared__ double data[];
    int blkID = blockIdx.x;
    int trdID = blkID * blockDim.x + threadIdx.x;
    int i, j, row_element_index;
    double total;
    //row_element_index = threadIdx.x * workload;
    i = threadIdx.x;
    while(i < n)
        {
        data[i] = d_samples[i * dim + blkID];
        i = i + num_of_threads;
        }

    __syncthreads();

    i = threadIdx.x;
    while(i < dim)
        {
        j = 0;
        total = 0;
        while(j < n)
            {
            total = total + (data[j] - d_mean[blkID]) * (d_samples[j * dim + i] - d_mean[i]);
            ++j;
            }
        d_scatter[blkID * dim + i] = total;
        i = i + num_of_threads;
        }
    }

__global__ static void kernel_compute_scatter_matrix_seq_col_maj(double *d_scatter, const double *d_mean, const double *d_samples, const int workload, const int dim, const int n)
    {
    int blkID = blockIdx.x; //Fixed
    int trdID = blkID * blockDim.x + threadIdx.x;
    int i, j, col_element_index;
    double total;
    col_element_index = threadIdx.x * workload; // Index of first element

    i = 0;
    while(i < workload)
        {
        if(col_element_index >= dim)
            break;
        else
            {
            //printf("(%d, %d)\n", blkID, row_element_index);
            total = 0;
            j = 0;
            while(j < n)
                {
                total = total + (d_samples[j * dim + col_element_index] - d_mean[col_element_index]) * (d_samples[j * dim + blkID] - d_mean[blkID]);
                ++j;
                }
            d_scatter[col_element_index * dim + blkID] = total;
            }
        ++col_element_index;
        ++i;
        }
    }

__global__ static void kernel_compute_scatter_matrix_seq_col_maj_memory_sharing(double *d_scatter, const double *d_mean, const double *d_samples, const int workload, const int dim, const int n)
    {
    extern __shared__ double data[];
    int blkID = blockIdx.x; //Fixed
    int trdID = blkID * blockDim.x + threadIdx.x;
    int i, j, col_element_index;
    double total;
    col_element_index = threadIdx.x * workload; // Index of first element

    i = threadIdx.x;
    while(i < n)
        {
        data[i] = d_samples[i * dim + blkID];
        i = i + num_of_threads;
        }

    __syncthreads();

    i = 0;
    while(i < workload)
        {
        if(col_element_index >= dim)
            break;
        else
            {
            //printf("(%d, %d)\n", blkID, row_element_index);
            total = 0;
            j = 0;
            while(j < n)
                {
                total = total + (d_samples[j * dim + col_element_index] - d_mean[col_element_index]) * (data[j] - d_mean[blkID]);
                ++j;
                }
            d_scatter[col_element_index * dim + blkID] = total;
            }
        ++col_element_index;
        ++i;
        }
    }

__global__ static void kernel_compute_scatter_matrix_interleve_col_maj(double *d_scatter, const double *d_mean, const double *d_samples, const int workload, const int dim, const int n)
    {
    int blkID = blockIdx.x; //Fixed
    int trdID = blkID * blockDim.x + threadIdx.x;
    int i, j, col_element_index;
    double total;
    col_element_index = threadIdx.x * workload; // Index of first element

    i = threadIdx.x;
    while(i < dim)
        {
        j = 0;
        total = 0;
        while(j < n)
            {
            total = total + (d_samples[j * dim + i] - d_mean[i]) * (d_samples[j * dim + blkID] - d_mean[blkID]);
            ++j;
            }
        d_scatter[i * dim + blkID] = total;
        i = i + num_of_threads;
        }
    }

__global__ static void kernel_compute_scatter_matrix_interleve_col_maj_memory_sharing(double *d_scatter, const double *d_mean, const double *d_samples, const int workload, const int dim, const int n)
    {
    extern __shared__ double data[];
    int blkID = blockIdx.x; //Fixed
    int trdID = blkID * blockDim.x + threadIdx.x;
    int i, j, col_element_index;
    double total;
    col_element_index = threadIdx.x * workload; // Index of first element

    i = threadIdx.x;
    while(i < n)
        {
        data[i] = d_samples[i * dim + blkID];
        i = i + num_of_threads;
        }

    __syncthreads();

    i = threadIdx.x;
    while(i < dim)
        {
        j = 0;
        total = 0;
        while(j < n)
            {
            total = total + (d_samples[j * dim + i] - d_mean[i]) * (data[j] - d_mean[blkID]);
            ++j;
            }
        d_scatter[i * dim + blkID] = total;
        i = i + num_of_threads;
        }
    }

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
    {
    runTest( argc, argv);
	//system("PAUSE");
    return EXIT_SUCCESS;
    }

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test
////////////////////////////////////////////////////////////////////////////////
void runTest( int argc, char** argv)
    {
    hipError_t err = hipSuccess;
    int n,dim;  //n = number   dim = dimension   command:./exe n dim
    int num_of_blocks, workload;
	double *samples, *scatter, *mean, *eigenvalue, *eigenvector, *wmatrix, *result;
    double *d_samples, *d_scatter, *d_mean;

    clock_gettime(CLOCK_REALTIME, &start_time);
	
    if (argc == 3)
        {
		n = atoi(argv[1]);
		dim = atoi(argv[2]);
        }
    else
        {
    	n=40;
		dim=3;
        }

	samples = new double [n * dim];

	if (!samples)
		fprintf(stderr, "error: can not allocate memory");


	//initialize random seed
    srand ( 7 );

    for( int i=0; i< n ; i++)
        {
		for(int j=0; j< dim ;j++)
            samples[i*dim + j] = rand() % 20 + 1;
        }

    //calculate the mean of each column 
    mean = new double [dim];

    for(int i=0; i < dim ;i++)
        mean[i]=0.0;

    for( int i=0; i< n ; i++)
        {
		for(int j=0; j< dim ;j++)
            mean[j] = mean[j] + samples[i*dim + j];
        }

	for(int i=0; i < dim ;i++)
        mean[i] = mean[i] / n;

    scatter = new double [dim*dim];// compute Scatter Matrix

    clock_gettime(CLOCK_REALTIME, &compute_scatter);

    err = hipMalloc(&d_scatter, sizeof(double) * dim * dim);
    if (err != hipSuccess)
        {
        printf("Fail to malloc &ac. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    err = hipMalloc(&d_samples, sizeof(double) * n * dim);
    if (err != hipSuccess)
        {
        printf("Fail to malloc &ac. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    err = hipMalloc(&d_mean, sizeof(double) * dim);
    if (err != hipSuccess)
        {
        printf("Fail to malloc &ac. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

    err = hipMemcpy(d_samples, samples, n * dim * sizeof(double),hipMemcpyHostToDevice);
    if (err != hipSuccess)
        {
        printf("Fail to malloc &ac. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    err = hipMemcpy(d_mean, mean, dim * sizeof(double),hipMemcpyHostToDevice);
    if (err != hipSuccess)
        {
        printf("Fail to malloc &ac. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

    num_of_blocks = dim;
    workload = (dim-1)/128 + 1;
    //kernel_compute_scatter_matrix_seq<<<num_of_blocks, num_of_threads>>>(d_scatter, d_mean, d_samples, workload, dim, n);
    //kernel_compute_scatter_matrix_seq_memory_sharing<<<num_of_blocks, num_of_threads, sizeof(double) * n>>>(d_scatter, d_mean, d_samples, workload, dim, n);
    //kernel_compute_scatter_matrix_interleve<<<num_of_blocks, num_of_threads>>>(d_scatter, d_mean, d_samples, workload, dim, n);
    kernel_compute_scatter_matrix_interleve_memory_sharing<<<num_of_blocks, num_of_threads, sizeof(double) * n>>>(d_scatter, d_mean, d_samples, workload, dim, n);
    //kernel_compute_scatter_matrix_seq_col_maj<<<num_of_blocks, num_of_threads>>>(d_scatter, d_mean, d_samples, workload, dim, n);
    //kernel_compute_scatter_matrix_seq_col_maj_memory_sharing<<<num_of_blocks, num_of_threads, sizeof(double) * n>>>(d_scatter, d_mean, d_samples, workload, dim, n);
    //kernel_compute_scatter_matrix_interleve_col_maj<<<num_of_blocks, num_of_threads>>>(d_scatter, d_mean, d_samples, workload, dim, n);
    //kernel_compute_scatter_matrix_interleve_col_maj_memory_sharing<<<num_of_blocks, num_of_threads, sizeof(double) * n>>>(d_scatter, d_mean, d_samples, workload, dim, n);
    err = hipGetLastError();
    if (err != hipSuccess)
        {
        printf("Fail to lauch the kernel. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }

    err = hipMemcpy(scatter, d_scatter, dim * dim * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        {
        printf("Fail to malloc &ac. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    err = hipFree(d_scatter);
    if (err != hipSuccess)
        {
        printf("Fail to malloc &ac. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    err = hipFree(d_samples);
    if (err != hipSuccess)
        {
        printf("Fail to malloc &ac. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    err = hipFree(d_mean);
    if (err != hipSuccess)
        {
        printf("Fail to malloc &ac. Error code: %s. \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    //check
    /*cout << endl << "********** Scatter Matrix **********" << endl;
	for( int i=0; i< dim ; i++)
        {
		for(int j=0; j< dim ;j++)
            {
            cout << scatter[i*dim+j]<<"  ";
            }
        cout << endl;
	    }*/

	eigenvalue = new double [dim];
	eigenvector = new double [dim*dim];

	clock_gettime(CLOCK_REALTIME, &before_jacobi);
    //cout << "Before Jacobi" << endl;
    Jacobi(scatter, dim, eigenvector, eigenvalue, 0.01, 20); // compute eigenvectors + eigenvalues + sort
    //cout << "After Jacobi" << endl;
    clock_gettime(CLOCK_REALTIME, &after_jacobi);
    //check
    /*cout << endl << "********** Eigenvector **********" << endl;
	for( int i=0; i< dim ; i++)
        {
		for(int j=0; j< dim ;j++)
            {
            cout << eigenvector[i*dim+j] << "  ";
            }
        cout << endl;
        }

    cout << endl << "********** Eigenvalue **********" << endl;
	for( int i=0; i< dim ; i++)
        {
		cout << eigenvalue[i] << "  ";
        }*/

	wmatrix = new double [2*dim];   // find the biggest 2 eigenvectors

	for(int i=0 ; i<dim ; i++)
        {
		wmatrix[i]=eigenvector[i*dim];
		wmatrix[dim+i]=eigenvector[i*dim+1];
        }

    result = new double [2*n];  //result

	for(int i=0 ; i<n ; i++)
        {
		result[i]=0;
		result[i+n]=0;

		for(int j=0 ; j<dim ;j++)
            {
			result[i]+= wmatrix[j]*samples[i*dim + j];
			result[i+n]+= wmatrix[j+dim]*samples[i*dim + j];
            }
		//cout<<result[i]<<" "<<result[i+n]<<endl;    //check
        }

    clock_gettime(CLOCK_REALTIME, &end_time);


    printf("s_time.tv_sec:%ld, s_time.tv_nsec:%09ld\n", start_time.tv_sec, start_time.tv_nsec);
    printf("e_time.tv_sec:%ld, e_time.tv_nsec:%09ld\n", end_time.tv_sec, end_time.tv_nsec);
    if(end_time.tv_nsec > start_time.tv_nsec)
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        end_time.tv_sec - start_time.tv_sec,
        end_time.tv_nsec - start_time.tv_nsec);
        }
    else
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        end_time.tv_sec - start_time.tv_sec - 1,
        end_time.tv_nsec - start_time.tv_nsec + 1000*1000*1000);
        }
    
    cout << endl << "Time breakdown: " << endl;
    cout << "========== Front Part ==========" << endl;
    printf("s_time.tv_sec:%ld, s_time.tv_nsec:%09ld\n", start_time.tv_sec, start_time.tv_nsec);
    printf("e_time.tv_sec:%ld, e_time.tv_nsec:%09ld\n", compute_scatter.tv_sec, compute_scatter.tv_nsec);
    if(compute_scatter.tv_nsec > start_time.tv_nsec)
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        compute_scatter.tv_sec - start_time.tv_sec,
        compute_scatter.tv_nsec - start_time.tv_nsec);
        }
    else
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        compute_scatter.tv_sec - start_time.tv_sec - 1,
        compute_scatter.tv_nsec - start_time.tv_nsec + 1000*1000*1000);
        }

    cout << endl << "========== Computing Scatter Matrix ==========" << endl;
    printf("s_time.tv_sec:%ld, s_time.tv_nsec:%09ld\n", compute_scatter.tv_sec, compute_scatter.tv_nsec);
    printf("e_time.tv_sec:%ld, e_time.tv_nsec:%09ld\n", before_jacobi.tv_sec, before_jacobi.tv_nsec);
    if(before_jacobi.tv_nsec > compute_scatter.tv_nsec)
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        before_jacobi.tv_sec - compute_scatter.tv_sec,
        before_jacobi.tv_nsec - compute_scatter.tv_nsec);
        }
    else
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        before_jacobi.tv_sec - compute_scatter.tv_sec - 1,
        before_jacobi.tv_nsec - compute_scatter.tv_nsec + 1000*1000*1000);
        }

    cout << endl << "========== Jacobi ==========" << endl;
    printf("s_time.tv_sec:%ld, s_time.tv_nsec:%09ld\n", before_jacobi.tv_sec, before_jacobi.tv_nsec);
    printf("e_time.tv_sec:%ld, e_time.tv_nsec:%09ld\n", after_jacobi.tv_sec, after_jacobi.tv_nsec);
    if(after_jacobi.tv_nsec > before_jacobi.tv_nsec)
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        after_jacobi.tv_sec - before_jacobi.tv_sec,
        after_jacobi.tv_nsec - before_jacobi.tv_nsec);
        }
    else
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        after_jacobi.tv_sec - before_jacobi.tv_sec - 1,
        after_jacobi.tv_nsec - before_jacobi.tv_nsec + 1000*1000*1000);
        }

    cout << endl << "========== After Jacobi ==========" << endl;
    printf("s_time.tv_sec:%ld, s_time.tv_nsec:%09ld\n", after_jacobi.tv_sec, after_jacobi.tv_nsec);
    printf("e_time.tv_sec:%ld, e_time.tv_nsec:%09ld\n", end_time.tv_sec, end_time.tv_nsec);
    if(end_time.tv_nsec > after_jacobi.tv_nsec)
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        end_time.tv_sec - after_jacobi.tv_sec,
        end_time.tv_nsec - after_jacobi.tv_nsec);
        }
    else
        {
        printf("[diff_time:%ld.%09ld sec]\n",
        end_time.tv_sec - after_jacobi.tv_sec - 1,
        end_time.tv_nsec - after_jacobi.tv_nsec + 1000*1000*1000);
        }

    cout << endl << "Waiting for file output......" << endl;
    
    ofstream fout;
    fout.open("result_CUDA.txt");

    for(int i=0 ; i < 2 * n ; i++)
        {
        fout << result[i] << " ";    //check
        }

    fout.close();
  //

#define TRACEBACK
#ifdef TRACEBACK

/*	FILE *fpo = fopen("result.txt","w");
	fprintf(fpo, "print traceback value GPU:\n");

	for (int i = max_cols- 2,  j = max_cols - 2; i>=0, j>=0;){
		int nw, n, w, traceback;
		if ( i == max_cols - 2 && j == max_cols - 2 )
			fprintf(fpo, "%d ", input_itemsets[ i * max_cols + j]); //print the first element
		if ( i == 0 && j == 0 )
           break;
		if ( i > 0 && j > 0 ){
			nw = input_itemsets[(i - 1) * max_cols + j - 1];
		    w  = input_itemsets[ i * max_cols + j - 1 ];
            n  = input_itemsets[(i - 1) * max_cols + j];
		}
		else if ( i == 0 ){
		    nw = n = LIMIT;
		    w  = input_itemsets[ i * max_cols + j - 1 ];
		}
		else if ( j == 0 ){
		    nw = w = LIMIT;
            n  = input_itemsets[(i - 1) * max_cols + j];
		}
		else{
		}

		//traceback = maximum(nw, w, n);
		int new_nw, new_w, new_n;
		new_nw = nw + referrence[i * max_cols + j];
		new_w = w - penalty;
		new_n = n - penalty;

		traceback = maximum(new_nw, new_w, new_n);
		if(traceback == new_nw)
			traceback = nw;
		if(traceback == new_w)
			traceback = w;
		if(traceback == new_n)
            traceback = n;

		fprintf(fpo, "%d ", traceback);

		if(traceback == nw )
		{i--; j--; continue;}

        else if(traceback == w )
		{j--; continue;}

        else if(traceback == n )
		{i--; continue;}

		else
		;
	}

	fclose(fpo);
*/
#endif

	delete scatter,eigenvalue,eigenvector,wmatrix,result,mean;


}

